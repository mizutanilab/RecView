#include "hip/hip_runtime.h"
#if !defined( _CUDARECONST_CU_ )
#define _CUDARECONST_CU_

#include <math.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
//#include "cudaFFT.cu"

////////////////////////
//Tomographic reconstruction routines
typedef int BOOL;

#include "reconstinfo.h"
#include "cudaReconst.h"
#include "constants.h"
//#include "sm_12_atomic_functions.h"

texture<int, 1, hipReadModeElementType> tex_igp;
int blocksize = CUDA_BLOCKSIZE;

__global__ void
projKernel9(int* d_ifp, int ixdimp, float fsin, float fcos, float foffset) {
	int ix = blockDim.x * blockIdx.x + threadIdx.x;
	if (ix >= ixdimp) return;
	int iy = blockIdx.y;
	float fx1 = ix * fcos + iy * fsin + foffset;
	int ixy = ix + ixdimp * iy;
	d_ifp[ixy] += tex1Dfetch(tex_igp, (int)(fx1));
}

__global__ void
projAtomicKernel9(int* d_ifp, int ixdimp, float fsin, float fcos, float foffset) {
	int ix = blockDim.x * blockIdx.x + threadIdx.x;
	if (ix >= ixdimp) return;
	int iy = blockIdx.y;
	float fx1 = ix * fcos + iy * fsin + foffset;
	int ixy = ix + ixdimp * iy;
	atomicAdd( &(d_ifp[ixy]), tex1Dfetch(tex_igp, (int)(fx1)) );
}

/*
__global__ void
projKernel9b(int* d_ifp, int ixdimp, float fsin, float fcos, float foffset) {
	int ix = blockDim.x * blockIdx.x + threadIdx.x;
	if (ix >= ixdimp) return;
	foffset += ix * fcos;
	for (int iy = 0; iy < ixdimp; iy++) {
		int ix1 = (int)(iy * fsin + foffset);
		int ixy = ix + ixdimp * iy;
		d_ifp[ixy] += tex1Dfetch(tex_igp, ix1);
	}
}

__global__ void
projKernel10(int* d_ifp, int* d_igp, int ixdimp, float* d_fcos, float* d_fsin, float fcenter, int iSinoDimX, int iSinoDimY) {
	//prepare shared mem
	extern __shared__ float s_fcos[];
	float* s_fsin = &(s_fcos[iSinoDimY]);
	const int icpyblock = (iSinoDimY + blockDim.x - 1) / blockDim.x;
	const int icpyfrom = threadIdx.x * icpyblock;
	const int icpyto = (threadIdx.x + 1) * icpyblock < iSinoDimY ? (threadIdx.x + 1) * icpyblock : iSinoDimY;
	for (int i = icpyfrom; i < icpyto; i++) {
		s_fcos[i] = d_fcos[i];
		s_fsin[i] = d_fsin[i];
	}
	__syncthreads();
	//params
	int ix = blockDim.x * blockIdx.x + threadIdx.x;
	if (ix >= ixdimp) return;
	int iy = blockIdx.y;
	const int ixy = ix + ixdimp * iy;
	const int ixdimh = ixdimp / 2;
	if ((ix - ixdimh)*(ix - ixdimh) + (iy - ixdimh)*(iy - ixdimh) > ixdimh * ixdimh) return;
	//ix can be ix - ixdimh though this will change pixel values
	//ix -= ixdimh;
	//iy -= ixdimh;
	//if (ix * ix + iy * iy > ixdimh * ixdimh) return;
	//pixel sum
	int isum = 0;
	for (int i = 0; i < iSinoDimY; i++) {
		const float foffset = fcenter - ixdimh * (s_fcos[i] + s_fsin[i]);
		float fx1 = ix * s_fcos[i] + iy * s_fsin[i] + foffset;
		if ((fx1 >= 0)&&(fx1 < iSinoDimX)) isum += d_igp[(int)fx1 + iSinoDimX * i];
		//int ix1 = (int)(ix * d_fcos[i] + iy * d_fsin[i] + fcenter);//this will change pixel values
		//if ((ix1 >= 0)&&(ix1 < iSinoDimX)) isum += d_igp[ix1 + iSinoDimX * i];
	}
	d_ifp[ixy] = isum;
}
*/

//gazo.cpp, cudaReconstHost.cpp, and DlgProperty.cpp should also be rebuilt to switch on and off CUDAFFT.
#ifdef CUDAFFT
__global__ void
p2igpKernel(float2* d_p, int* d_igp, int ixdimp, int ihoffset, float fscale) {
	int ix = blockDim.x * blockIdx.x + threadIdx.x;
	if (ix >= ixdimp) return;
	int iy = blockIdx.y;
	//const int gidx = ix * DBPT_GINTP;
	const int gidx = ix * DBPT_GINTP;
	const int pidx = ix + ihoffset;
	const float p0 = d_p[pidx].x * fscale;
	const float p1p0 = (ix == ixdimp -1)? 0.0f : (d_p[pidx + 1].x * fscale - p0) / DBPT_GINTP;
	d_igp[gidx + iy] = (int)(p0 + p1p0 * iy);
	//
	//for (int j=0; j<ixdimp; j++) {
	//	const TCmpElmnt p0 = p[j + ihoffset].re * BACKPROJ_SCALE;
	//	const TCmpElmnt p1p0 = (j == ixdimp -1)? 
	//		0.0f : (p[j + ihoffset + 1].re - p[j + ihoffset + 1].re) / DBPT_GINTP * BACKPROJ_SCALE;
	//	const int gidx = (j + imargin) * DBPT_GINTP;
	//	for (int k=0; k<DBPT_GINTP; k++) {igp[gidx + k] = (int)(p0 + p1p0 * k);}
	//}
}

__global__ void
filtKernel(float2* d_p, float* d_filt, int ndim) {
	int ix = blockDim.x * blockIdx.x + threadIdx.x;
	if (ix >= ndim) return;
	d_p[ix].x *= d_filt[ix];
	d_p[ix].y *= d_filt[ix];
	//for (int k=0; k<ndim; k++) {d_p[k].x *= d_filt[k]; d_p[k].y *= d_filt[k];}
}

__global__ void
intpKernel(float2* d_p, short* d_strip, int ixdim, int ndim, int iIntpDim, float center) {
	int ix = blockDim.x * blockIdx.x + threadIdx.x;
	if (ix >= ixdim) return;
	int iy = blockIdx.y;
	int idx = (ix - (int)center) * iIntpDim + (ndim / 2) + iy;
	if (idx < 0) return;
	if (idx >= ndim) return;
	if (iy == 0) { d_p[idx].x = d_strip[ix]; return; }
	//interpolation
	if (ix == ixdim - 1) return;
	d_p[idx].x = (float)(d_strip[ix] * (iIntpDim - iy) + d_strip[ix + 1] * iy) / iIntpDim;
//190120
//	int idx = (ix - (int)center) * iIntpDim + (ndim / 2);
//	if (idx < 0) return;
//	if (idx >= ndim) return;
//	int iy = blockIdx.y;
//	if (iy == 0) {d_p[idx].x = d_strip[ix]; return;}
//	//interpolation
//	if (ix == ixdim - 1) return;
//	d_p[idx+iy].x = (float)(d_strip[ix] * (iIntpDim - iy) + d_strip[ix+1] * iy) / iIntpDim;
	/*
	for (int k=0; k<ixdim; k++) {
		int idx = (k - (int)center) * iIntpDim + (ndim / 2);
		if (idx < 0) continue;
		if (idx >= ndim) break;
		d_p[idx].x = d_strip[k];
		//interpolation
		if (k == ixdim - 1) break;
		for (int j=1; j<iIntpDim; j++) {
			d_p[idx+j].x = (float)(d_strip[k] * (iIntpDim - j) / iIntpDim + d_strip[k+1] * j / iIntpDim);
		}
	}*/
}

extern "C" 
void CudaDeconv(int ixdim, int iIntpDim, int ndim, float center,   
				float* d_filt, short* d_strip, int* d_igp, float2* d_p, hipfftHandle* fftplan) {
	//constants
	const int ixdimp = ixdim * iIntpDim;
	const int ixdimh = ixdimp / 2;
	const int ihoffset = ndim / 2 - ixdimh;
	//
	//kernel parameters
	//const int blocksize = CUDA_BLOCKSIZE;// ==> blockDim.x;
    dim3 dimBlock(blocksize, 1);
	const int gridsize = (int)(ceil( ixdimp / (float)blocksize));
	dim3 dimGrid_ixdimp(gridsize);//0<=blockIdx.x<gridsize
	//
	//interpolation
    const unsigned int mem_size_p = sizeof(float2) * ndim;
	hipMemset(d_p, 0, mem_size_p);
	//130923 cutilSafeCall(hipMemset(d_p, 0, mem_size_p) );
	//090312 hipMemset(d_p, 0, mem_size_p);
	const int gridsize_intp = (int)(ceil( ixdim / (float)blocksize));
	dim3 dimGrid_intp(gridsize_intp, iIntpDim);
	intpKernel<<< dimGrid_intp, dimBlock >>>(d_p, d_strip, ixdim, ndim, iIntpDim, center);
	//FFT-filter
	hipfftExecC2C(*fftplan, (hipfftComplex*)d_p, (hipfftComplex*)d_p, HIPFFT_FORWARD );
	//
	const int gridsize_ndim = (int)(ceil( ndim / (float)blocksize));
	dim3 dimGrid_ndim(gridsize_ndim);
	filtKernel<<< dimGrid_ndim, dimBlock >>>(d_p, d_filt, ndim);
	//
	hipfftExecC2C(*fftplan, (hipfftComplex*)d_p, (hipfftComplex*)d_p, HIPFFT_BACKWARD );
	//
	float fscale = (float)BACKPROJ_SCALE / ndim;
	dim3 dimGrid_p2igp(gridsize, DBPT_GINTP);
	p2igpKernel<<< dimGrid_p2igp, dimBlock >>>(d_p, d_igp, ixdimp, ihoffset, fscale);
}
#endif

extern "C"
void CudaBackProjStream(int ixdimp, float center, int iCenterOffset, int iIntpDim, double theta, int* d_ifp, int* d_igp, hipStream_t stream) {
	//constants
	//const int ixdimp = ixdim * iIntpDim;
	const int ixdimh = ixdimp / 2;
	//
	//kernel parameters
	//const int blocksize = CUDA_BLOCKSIZE;// ==> blockDim.x;
	dim3 dimBlock(blocksize, 1);
	const int gridsize = (int)(ceil(ixdimp / (float)blocksize));
	//igp texture
	textureReference* texRefPtr;
	if (hipSuccess != hipGetTextureReference((const textureReference **)&texRefPtr, &tex_igp)) return;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int>();
	tex_igp.addressMode[0] = hipAddressModeClamp;
	tex_igp.filterMode = hipFilterModePoint;
	tex_igp.normalized = false;    // access with integer texture coordinates
	const unsigned int mem_size_igp = sizeof(int) * ixdimp * DBPT_GINTP;
	//
	if (hipSuccess != hipBindTexture(0, texRefPtr, d_igp, &channelDesc, mem_size_igp)) return;
	//params
	const float fcos = (float)(cos(theta) * DBPT_GINTP);
	const float fsin = (float)(-sin(theta) * DBPT_GINTP);
	//210105 const float fcenter = (float)((ixdimh + center - (int)(center)) * DBPT_GINTP);
	//a possible revision related to the bug fix of 210105 in the corresponding intel routine
	const float fcenter = (iIntpDim == 1) ? (ixdimh + center - (int)(center)) * DBPT_GINTP :
											(ixdimh + (center - (int)(center)) * iIntpDim) * DBPT_GINTP;
	const float foffset = fcenter - ixdimh * (fcos + fsin) + DBPT_GINTP * iCenterOffset;
	//Kernel
	int iydim = ixdimp;
	dim3 dimGrid(gridsize, iydim);// 0<=blockIdx.x<gridsize, 0<=blockIdx.y<iydim
	projAtomicKernel9 << < dimGrid, dimBlock, 0, stream >> > (d_ifp, ixdimp, fsin, fcos, foffset);//no delays
	//projKernel9 << < dimGrid, dimBlock, 0, stream >> > (d_ifp, ixdimp, fsin, fcos, foffset);
}

__global__ void
px2igpKernelStream(float* d_px, int* d_igp, int ixdimp) {
	int ix = blockDim.x * blockIdx.x + threadIdx.x;
	if (ix >= ixdimp) return;
	const int iy = blockIdx.y;
	const int gidx = ix * DBPT_GINTP;
	//const int pidx = (ix + ihoffset < 0) ? 0 : ((ix + ihoffset >= ndim)? ndim-1 : ix + ihoffset);
	const float p0 = d_px[ix] * BACKPROJ_SCALE;
	const float p1p0 = (ix == ixdimp - 1) ? 0.0f : (d_px[ix + 1] * BACKPROJ_SCALE - p0) / DBPT_GINTP;
	d_igp[gidx + iy] = (int)(p0 + p1p0 * iy);
}
extern "C" void CudaSinoPx2igpStream(int ixdimp, int* d_igp, float* d_px, hipStream_t stream) {
	dim3 dimBlock(blocksize, 1);
	const int gridsize = (int)(ceil(ixdimp / (float)blocksize));
	dim3 dimGrid_p2igp(gridsize, DBPT_GINTP);
	px2igpKernelStream << < dimGrid_p2igp, dimBlock, 0, stream >> > (d_px, d_igp, ixdimp);
}

/*
__global__ void
px2igpKernel(float* d_px, int* d_igp, int ixdimp, int ihoffset, int ndim, int igpdimx) {
	int ix = blockDim.x * blockIdx.x + threadIdx.x;
	if (ix >= ixdimp) return;
	const int isino = blockIdx.z;
	const int iy = blockIdx.y;
	const int gidx = ix * DBPT_GINTP + isino * igpdimx;
	const int pidx = ix + ihoffset + isino * ndim;
	const float p0 = d_px[pidx] * BACKPROJ_SCALE;
	const float p1p0 = (ix == ixdimp - 1) ? 0.0f : (d_px[pidx + 1] * BACKPROJ_SCALE - p0) / DBPT_GINTP;//should be revised accroding to Stream version
	d_igp[gidx + iy] = (int)(p0 + p1p0 * iy);

//	for (int i = (ri->iStartSino); i < (ri->iLenSinogr - 1); i += (ri->iStepSino)) {
//		int isino = (i - (ri->iStartSino)) / (ri->iStepSino);
//		if (!(ri->bInc[i] & CGAZODOC_BINC_SAMPLE)) continue;
//		for (int j = 0; j < ixdimp; j++) {
//			const TCmpElmnt p0 = h_px[j + ihoffset + isino * ndim] * BACKPROJ_SCALE;
//			const TCmpElmnt p1p0 = (j == ixdimp - 1) ?
//				0.0f : (h_px[j + ihoffset + 1 + isino * ndim] - h_px[j + ihoffset + isino * ndim]) / DBPT_GINTP * BACKPROJ_SCALE;
//			const int gidx = (j + imargin) * DBPT_GINTP + isino * igpdimx;
//			for (int k = 0; k < DBPT_GINTP; k++) { igp[gidx + k] = (int)(p0 + p1p0 * k); }
//		}
//	}
}

extern "C" void CudaSinoPx2igp(int ndim, int ixdimp, int igpdimx, int igpdimy, int* d_igp, float* d_px) {
	const int ixdimh = ixdimp / 2;
	const int ihoffset = ndim / 2 - 1 - ixdimh;
	dim3 dimBlock(blocksize, 1, 1);
	const int gridsize = (int)(ceil(ixdimp / (float)blocksize));
	dim3 dimGrid_p2igp(gridsize, DBPT_GINTP, igpdimy);
	px2igpKernel << < dimGrid_p2igp, dimBlock >> > (d_px, d_igp, ixdimp, ihoffset, ndim, igpdimx);
}

extern "C" 
void CudaBackProj3(int ixdim, int iIntpDim, float center, int iStartSino, int iLenSinogr, int iStepSino, float* pfdeg, float ftilt, int* d_ifp, int* d_igp) {
	//constants
	const int ixdimp = ixdim * iIntpDim;
	const int ixdimh = ixdimp / 2;
	const int imargin = 0;
	const int igpdimx = (ixdimp + imargin * 2) * DBPT_GINTP;
	//
	//kernel parameters
	//const int blocksize = CUDA_BLOCKSIZE;// ==> blockDim.x;
    dim3 dimBlock(blocksize, 1);
	const int gridsize = (int)(ceil( ixdimp / (float)blocksize));
	//igp texture
	textureReference* texRefPtr;
	if (hipSuccess != hipGetTextureReference((const textureReference **)&texRefPtr, &tex_igp)) return;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int>();
	tex_igp.addressMode[0] = hipAddressModeClamp;
	tex_igp.filterMode = hipFilterModePoint;
	tex_igp.normalized = false;    // access with integer texture coordinates
	const unsigned int mem_size_igp = sizeof(int) * ixdimp * DBPT_GINTP;
	//
	for (int i = iStartSino; i < iLenSinogr - 1; i += iStepSino) {
		const int isino = (i - iStartSino) / iStepSino;
		const double theta = (pfdeg[i] + ftilt) * DEG_TO_RAD; 

		if (hipSuccess != hipBindTexture(0, texRefPtr, &(d_igp[isino * igpdimx]), &channelDesc, mem_size_igp)) return;
		//params
		const float fcos = (float)(cos(theta) * DBPT_GINTP);
		const float fsin = (float)(-sin(theta) * DBPT_GINTP);
		const float fcenter = (float)((ixdimh + center - (int)(center)) * DBPT_GINTP);
		const float foffset = fcenter - ixdimh * (fcos + fsin);
		//Kernel
		int iydim = ixdimp;
		dim3 dimGrid(gridsize, iydim);// 0<=blockIdx.x<gridsize, 0<=blockIdx.y<iydim
		projKernel9 << < dimGrid, dimBlock >> > (d_ifp, ixdimp, fsin, fcos, foffset);
		//Kernel9b: slow
		//dim3 dimGrid9b(gridsize, 1);// 0<=blockIdx.x<gridsize
		//projKernel9b << < dimGrid9b, dimBlock >> > (d_ifp, ixdimp, fsin, fcos, foffset);
	}

	hipUnbindTexture(tex_igp);
}

//190101
extern "C"
void CudaBackProj2(int ixdim, int iIntpDim, float center, int iSinoDimX, int iSinoDimY, int* d_ifp, int* d_igp, float* d_fcos, float* d_fsin) {
	//constants
	const int ixdimp = ixdim * iIntpDim;
	const int ixdimh = ixdimp / 2;
	//
	//const int iSinoDimX = (ixdimp + imargin * 2) * DBPT_GINTP;
	//const int iSinoDimY = ((ri->iLenSinogr - 1) - (ri->iStartSino) + (ri->iStepSino) - 1) / (ri->iStepSino);
	//
	//kernel parameters
	//const int blocksize = CUDA_BLOCKSIZE;// ==> blockDim.x;
	dim3 dimBlock(blocksize, 1);
	const int gridsize = (int)(ceil(ixdimp / (float)blocksize));
	//params
	const float fcenter = (float)((ixdimh + center - (int)(center)) * DBPT_GINTP);
	//Kernel
	const int iydim = ixdimp;
	const int shared_mem_size = iSinoDimY * sizeof(float) * 2;
	dim3 dimGrid(gridsize, iydim);// 0<=blockIdx.x<gridsize, 0<=blockIdx.y<iydim
	projKernel10 << < dimGrid, dimBlock, shared_mem_size >> > (d_ifp, d_igp, ixdimp, d_fcos, d_fsin, fcenter, iSinoDimX, iSinoDimY);
}

extern "C"
void CudaBackProj(int ixdim, int iIntpDim, float center, int iCenterOffset, double theta, int* d_ifp, int* d_igp) {
	//constants
	const int ixdimp = ixdim * iIntpDim;
	const int ixdimh = ixdimp / 2;
	//
	//kernel parameters
	//const int blocksize = CUDA_BLOCKSIZE;// ==> blockDim.x;
	dim3 dimBlock(blocksize, 1);
	const int gridsize = (int)(ceil(ixdimp / (float)blocksize));
	//igp texture
	textureReference* texRefPtr;
	if (hipSuccess != hipGetTextureReference((const textureReference **)&texRefPtr, &tex_igp)) return;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int>();
	tex_igp.addressMode[0] = hipAddressModeClamp;
	tex_igp.filterMode = hipFilterModePoint;
	tex_igp.normalized = false;    // access with integer texture coordinates
	const unsigned int mem_size_igp = sizeof(int) * ixdimp * DBPT_GINTP;
	//
	if (hipSuccess != hipBindTexture(0, texRefPtr, d_igp, &channelDesc, mem_size_igp)) return;
	//params
	const float fcos = (float)(cos(theta) * DBPT_GINTP);
	const float fsin = (float)(-sin(theta) * DBPT_GINTP);
	const float fcenter = (float)((ixdimh + center - (int)(center)) * DBPT_GINTP);
	const float foffset = fcenter - ixdimh * (fcos + fsin) + DBPT_GINTP * iCenterOffset;
	//Kernel
	int iydim = ixdimp;
	dim3 dimGrid(gridsize, iydim);// 0<=blockIdx.x<gridsize, 0<=blockIdx.y<iydim
	projKernel9 << < dimGrid, dimBlock >> > (d_ifp, ixdimp, fsin, fcos, foffset);
	//Kernel9b: slow
	//dim3 dimGrid9b(gridsize, 1);// 0<=blockIdx.x<gridsize
	//projKernel9b << < dimGrid9b, dimBlock >> > (d_ifp, ixdimp, fsin, fcos, foffset);
}
*/

__global__ void
sinoKernel(short* d_Dark, short* d_Incident, short* d_Strip, int ixmul, float t0) {
	int ix = blockDim.x * blockIdx.x + threadIdx.x;
	if (ix >= ixmul) return;
	if (d_Incident[ix] <= 0) {d_Strip[ix] = 0; return;}
	int iSample = d_Strip[ix] - d_Dark[ix];
	if (iSample < SINOGRAM_PIXEL_MIN) {d_Strip[ix] = 0; return;}
	d_Strip[ix] = (short)(log(d_Incident[ix] / (double)iSample) * LOG_SCALE + 0.5);
	//for (int j=0; j<ixmul; j++) {
	//	int d_Incident = d_Incident0[j] + (int)((d_Incident1[j] - d_Incident0[j] + 0.5) * t0) - d_Dark[j];
	//	if (d_Incident <= 0) {d_Strip[j] = 0; continue;}
	//	int iSample = d_Strip[j] - d_Dark[j];
	//	if (iSample < SINOGRAM_PIXEL_MIN) {d_Strip[j] = 0; continue;}
	//	d_Strip[j] = (short)(log((double)d_Incident / (double)iSample) * LOG_SCALE + 0.5);
	//}
}

extern "C" 
void CudaSinogram(short* d_Strip, int ixmul, float t0) {
    dim3 dimBlock(blocksize, 1);
	const int gridsize_ixmul = (int)(ceil( ixmul / (float)blocksize));
	dim3 dimGrid_ixmul(gridsize_ixmul);
	short* d_Incident = &(d_Strip[ixmul]);
	short* d_Dark = &(d_Strip[ixmul * 2]);
	//normal
	sinoKernel<<< dimGrid_ixmul, dimBlock >>>(d_Dark, d_Incident, d_Strip, ixmul, t0);
}

__global__ void
lsqfitKernel(short* d_ref, short* d_qry, int ixref, int iyref, int ixqry, int iyqry, int ix, int iy, 
					unsigned __int64* d_result) {
	int jrx = blockDim.x * blockIdx.x + threadIdx.x;
	if (jrx >= ixref) return;
	const int jqx = jrx + ix;
	if ((jqx < 0)||(jqx >= ixqry)) return;
	//unsigned __int64 nlsqsum = 0;
	int nlsqsum = 0;
	unsigned __int64 ilsqsum = 0;
	for (int jry=0; jry<iyref; jry++) {
		const int jqy = jry + iy;
		if ((jqy < 0)||(jqy >= iyqry)) continue;
		int dr = d_ref[jry * ixref + jrx];
		if (dr == SHRT_MIN) continue;
		int dq = d_qry[jqy * ixqry + jqx];
		if (dq == SHRT_MIN) continue;
		//130207 unsigned __int64 idiff = dr - dq;
		__int64 idiff = dr - dq;
		ilsqsum += idiff * idiff;
		nlsqsum++;
	}
	d_result[jrx] = ilsqsum;
	d_result[jrx + ixref] = nlsqsum;
	//atomicAdd(&(d_result[0]), ilsqsum);
	//atomicAdd(&(d_result[1]), nlsqsum);
}

extern "C" 
void CudaLsqfit(short* d_ref, short* d_qry, int ixref, int iyref, int ixqry, int iyqry,
					int ix, int iy, unsigned __int64* d_result) {
    dim3 dimBlock(blocksize, 1);
	const int gridsize = (int)(ceil( ixref / (float)blocksize));
	dim3 dimGrid(gridsize);
	lsqfitKernel<<< dimGrid, dimBlock >>>(d_ref, d_qry, ixref, iyref, ixqry, iyqry, ix, iy, d_result);
}

extern "C" int GetCudaDeviceCount(int iMinComputeCapability) {
    int deviceCount;
	hipError_t cerr = hipGetDeviceCount(&deviceCount);
	if (cerr == hipErrorNoDevice) {
		return 0;
	} else if (cerr == hipErrorInsufficientDriver) {
		return CUDA_ERROR_INSUFFICIENT_DRIVER;
	} else if (cerr != hipSuccess) {
		return CUDA_ERROR_DEVICE_GETCOUNT;
	}
	//130923 cutilSafeCall(hipGetDeviceCount(&deviceCount));

    //detect virtual device or low "compute capability" 181226
	//const int iMinComputeCapability = __CUDA_ARCH__;
	//The compute capability number is set in the Project-Property-CUDA C/C++ page
	//minimum number for CUDA Tk 10.0 is compute_30 (__CUDA_ARCH__ = 300)
	for (int i=0; i<deviceCount; i++) {
		hipDeviceProp_t deviceProp;
		if (hipSuccess != hipGetDeviceProperties(&deviceProp, i)) { return i | CUDA_ERROR_DEVICE_GETPROPERTY; }
		if (deviceProp.major == 9999 && deviceProp.minor == 9999) { return i | CUDA_ERROR_VIRTUAL_DEVICE_DETECTED; }//virtual device
		else if (deviceProp.major * 100 + deviceProp.minor * 10 < iMinComputeCapability) { return i | CUDA_ERROR_INSUFFICIENT_COMPUTE_CAPABILITY; }//low "compute capability"
	}

//	if (deviceCount) {
//	    hipDeviceProp_t deviceProp;
//		if (hipSuccess != hipGetDeviceProperties(&deviceProp, 0)) return 0;
//		//130923 cutilSafeCall(hipGetDeviceProperties(&deviceProp, 0));
//	    if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
//			//comment out the follwing line to enable virtual device
//			deviceCount = 0;
//		}
//	}
    return deviceCount;
}

extern "C" int GetCudaDeviceName(int iDevice, char* pcName, int iszcName) {//181226
	const int isz = (iszcName < 256) ? iszcName : 256;
	hipDeviceProp_t deviceProp;
	if (hipSuccess != hipGetDeviceProperties(&deviceProp, iDevice)) { return CUDA_ERROR_DEVICE_GETPROPERTY; }
	if (deviceProp.major == 9999 && deviceProp.minor == 9999) { return CUDA_ERROR_VIRTUAL_DEVICE_DETECTED; }//virtual device
	for (int i = 0; i < isz; i++) {
		char c = deviceProp.name[i];
		pcName[i] = c;
		if (c == 0) break;
	}
	return 0;
}

extern "C" int GetCudaDeviceComputingCapability(int iDevice, int* piMajor, int* piMinor) {//181226
	hipDeviceProp_t deviceProp;
	if (hipSuccess != hipGetDeviceProperties(&deviceProp, iDevice)) { return CUDA_ERROR_DEVICE_GETPROPERTY; }
	if (deviceProp.major == 9999 && deviceProp.minor == 9999) { return CUDA_ERROR_VIRTUAL_DEVICE_DETECTED; }//virtual device
	if (piMajor) *piMajor = deviceProp.major;
	if (piMinor) *piMinor = deviceProp.minor;
	return 0;
}

extern "C" int GetCudaMaxThreadsPerBlock(int iDevice) {
	hipDeviceProp_t deviceProp;
	if (hipSuccess != hipGetDeviceProperties(&deviceProp, iDevice)) { return CUDA_ERROR_DEVICE_GETPROPERTY; }
	return deviceProp.maxThreadsPerBlock;
}

extern "C" int GetCudaWarpSize(int iDevice) {
	hipDeviceProp_t deviceProp;
	if (hipSuccess != hipGetDeviceProperties(&deviceProp, iDevice)) { return CUDA_ERROR_DEVICE_GETPROPERTY; }
	return deviceProp.warpSize;
}

extern "C" int GetCudaNumberOfCores(int iDevice, int* piCores, int* piProcessors) {//190110
	hipDeviceProp_t deviceProp;
	if (hipSuccess != hipGetDeviceProperties(&deviceProp, iDevice)) { return CUDA_ERROR_DEVICE_GETPROPERTY; }
	if (deviceProp.major == 9999 && deviceProp.minor == 9999) { return CUDA_ERROR_VIRTUAL_DEVICE_DETECTED; }//virtual device
	//220422 if (piCores) *piCores = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
	if (piCores) {//220422
		if ((deviceProp.major == 8) && (deviceProp.minor == 6)) *piCores = 128;//GA102-7 has 64 FP32 + 64 FP/INT32 = 128 cores per SM (NVIDIA Ampere GA102 GPU Architecture, 2021)
		else *piCores = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
	}
	if (piProcessors) *piProcessors = deviceProp.multiProcessorCount;
	return 0;
}

extern "C" int GetCudaClockRate(int iDevice, int* piClockRate, int* piMemRate) {//190115
	hipDeviceProp_t deviceProp;
	if (hipSuccess != hipGetDeviceProperties(&deviceProp, iDevice)) { return CUDA_ERROR_DEVICE_GETPROPERTY; }
	if (deviceProp.major == 9999 && deviceProp.minor == 9999) { return CUDA_ERROR_VIRTUAL_DEVICE_DETECTED; }//virtual device
	if (piClockRate) *piClockRate = deviceProp.clockRate;
	if (piMemRate) *piMemRate = deviceProp.memoryClockRate;
	return 0;
}

/*
deviceQuery.cu

There is 1 device supporting CUDA

Device 0: "Quadro FX 3700"
  Major revision number:                         1
  Minor revision number:                         1
  Total amount of global memory:                 536870912 bytes
  Number of multiprocessors:                     14
  Number of cores:                               112
  Total amount of constant memory:               65536 bytes
  Total amount of shared memory per block:       16384 bytes
  Total number of registers available per block: 8192
  Warp size:                                     32
  Maximum number of threads per block:           512
  Maximum sizes of each dimension of a block:    512 x 512 x 64
  Maximum sizes of each dimension of a grid:     65535 x 65535 x 1
  Maximum memory pitch:                          262144 bytes
  Texture alignment:                             256 bytes
  Clock rate:                                    1.25 GHz
  Concurrent copy and execution:                 No

Test PASSED
*/

#endif //_CUDARECONST_CU_